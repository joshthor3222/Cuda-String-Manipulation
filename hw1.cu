#include "hip/hip_runtime.h"
//
//  hw1.c
//
//
//  Created by Josh Chavez on 4/7/16.
//
//

#include <math.h>
#include <stdio.h>

extern "C" {
#include "hw1funs.h"
#include "timing.h"
}


__global__ void kernel(char *words, char *finalwords, unsigned long long int *ret, int *occurences, int numWords,
                       int wordSize, int loopNum) {

    extern __shared__ char s[];
    __shared__ int hash;
    __shared__ int occur;

    int ix = threadIdx.x;
    int idx = blockIdx.x*wordSize + threadIdx.x + (blockIdx.y * 65535 * wordSize); //CORRECT IDX!!


    if (idx < (numWords * wordSize)) {
        char a = words[idx];
        if(a >= 'a' && a <= 'z') {
            s[ix] = a;
        }
        else if(a >= 'A' && a <= 'Z') {
            a = a + 32;
            s[ix] = a;
        }
        else
            s[ix] = ' ';

        __syncthreads();


        if(ix > 0) {
            if((s[ix] == ' ' || s[ix - 1] == ' ') && ix < wordSize) {
                s[ix] = ' ';
                s[ix + 1] = ' ';
            }
        }

        //below this works!!
        if (threadIdx.x == 0) { //this thread adjusts the shared memory
            occur = 0;
            unsigned long long int res = 1;
            for (int i = 0; (s[i] != '\0' && s[i] != ' '); i++) {
                res = res * 31 + s[i];
            }
            hash = res % numWords;

            unsigned long long int old = atomicCAS(&ret[hash], 0, res); //global - slow, 1 call.
            while(old != 0) {
                if(old == res)
                    break;
                else {
                    if(hash < numWords)
                        hash++;
                    else
                        hash = 0;
                    old = atomicCAS(&ret[hash], 0, res);
                }
            }
            occur = atomicAdd(&occurences[hash], 1);
        }
        __syncthreads();
        if(occur == 0) {
            finalwords[(hash * wordSize) + ix] = s[ix];
        }
    }
}

//this will get the words in a array to be used by a cuda kernel.
void getCount(FILE *fin, int *words, int *wordLength) {
    char temp[100];
    int i = 0;
    while (fscanf(fin, "%s", temp) == 1) {
        if (isalpha(temp[0])) {
            i++;
            int m = strlen(temp);
            if (m > *wordLength) {
                *wordLength = m;
            }
        }
    }
    *words = i;
    rewind(fin);
}

void getWordsStart(FILE *fin, char **wordList) {
    char temp[100];
    int i = 0;
    while (fscanf(fin, "%s", temp) == 1) {
        if (isalpha(temp[0])) {
            int m = 0;
            while (isalpha(temp[m])) {
                temp[m] = tolower(temp[m]);
                m++;
            }
            temp[m] = '\0';
            wordList[i] = (char *) malloc((strlen(temp) + 1) * sizeof(char));
            strcpy(wordList[i], temp);
            i++;
        }
    }
}

void sequential(char **words, char **finalwords, unsigned long long int *ret, int *occurences, int numWords,
                int wordSize) {
    for (int j = 0; j < numWords; j++) {
        char newWord[wordSize];
        strcpy(newWord, words[j]);
        unsigned long long int res = 1; // Initialize result
        for (int i = 0; (newWord[i] != '\0' && newWord[i] != ' '); i++) {
            res = res * 31 + newWord[i];
        }


        //res is full value we are saving.

        int hash = res % numWords;
        while (ret[hash] != 0) {// && ret[hash] != res) {
            if (ret[hash] == res) {
                if (strcmp(finalwords[hash], newWord) == 0) {
                    occurences[hash] = occurences[hash] + 1;
                    break;
                }
            }
            if (hash < numWords)
                hash++;
            else
                hash = 0;
        }
        if (ret[hash] == 0) {
            ret[hash] = res;
            occurences[hash] = occurences[hash] + 1;
            finalwords[hash] = words[j];
        }
    }
}

int main() {
    hipSetDevice(0);

    double time, then, timea, timeb;

    //Sequential time, currently commented out.

    FILE *fin = NULL;
    FILE *fout;
    char **wordList;
    char **seqFinalList;


    int *wordsA, *wordLengthA;
    int aa = 0, ba = 0;
    wordsA = &aa;
    wordLengthA = &ba;

    fin = openFile();

    time = currentTime();

    getCount(fin, wordsA, wordLengthA); //get words


    wordList = (char **) malloc(*wordsA * sizeof(char *));
    getWordsStart(fin, wordList);

    seqFinalList = (char **) malloc(*wordsA * sizeof(char *));

    unsigned long long int *hashVals = (unsigned long long int *) malloc(*wordsA * sizeof(unsigned long long int));
    int *occurVals = (int *) malloc(*wordsA * sizeof(int));

    sequential(wordList, seqFinalList, hashVals, occurVals, *wordsA, *wordLengthA);
    //old wordlist goes into new seqfinallist so there is only one occurence of each word
    //hashvals is temporary for indexing, occurvals has number of occurences

    free(hashVals); //no longer needed, lose it.

    int newNum = 0;
    for (int z = 0; z < *wordsA; z++) {
        if (occurVals[z] != 0) {
            newNum++;
        }
    }
    //gets new size for everything

    int *seqOccurences = (int *) malloc(newNum * sizeof(int));
    char **seqWords = (char **) malloc(newNum * sizeof(char *));
    int next = 0;
    for (int z = 0; z < *wordsA; z++) {
        if (occurVals[z] != 0) {
            seqOccurences[next] = occurVals[z];
            seqWords[next] = seqFinalList[z];
            next++;
        }
    }
    free(occurVals);
    //puts occurences in new minimal array

    quick_sort(seqWords, seqOccurences, 0, newNum - 1); //works!


    fout = fopen("alphaOutput.txt", "w");
    cudaSaveList(seqWords, seqOccurences, newNum, fout);
    fclose(fout);
    printf("Alphabetical order saved in alphaOutput.txt\n");

    quickSortOccur(seqOccurences, seqWords, 0, newNum - 1); //works!

    FILE *fout1;
    fout1 = fopen("occurOutput.txt", "w");
    cudaSaveList(seqWords, seqOccurences, newNum, fout1);
    fclose(fout1);
    printf("Order of occurences saved in occurOutput.txt\n");


    for (int z = 0; z < *wordsA; z++) {
        free(wordList[z]); //gets rid of all words
    }
    free(wordList);
    free(seqOccurences);
    free(seqWords);
    free(seqFinalList);

    then = currentTime();

    printf("Sequential time spent: %lf seconds\n", then - time);
    //*/
    //Parallel execution







    timea = currentTime();
    rewind(fin);
    int *words;
    int a = 0;
    words = &a;
    int *wordLength;
    int b = 0;
    wordLength = &b;

    getCount(fin, words, wordLength); //gets total amount of words, and max word length;
    *wordLength = *wordLength + 1;

    int pitchSize = (*words * *wordLength);
    //do chunks
    char *list = (char *) malloc(pitchSize * sizeof(char));


    printf("\namount of words: %d; max length of a word: %d\n\n", *words, *wordLength);


    char temp[*wordLength];
    int i = 0;
    while (i < pitchSize) {
        if (fscanf(fin, "%s", temp) == 1) {
            if (isalpha(temp[0])) {
                for (int z = 0; z < *wordLength; z++) {
                    if (temp[z] != '\0') {
                        list[i] = temp[z];
                    }
                    else {
                        list[i] = ' ';
                    }
                    i++;

                }
            }
        }
    }
    char *dev_words;
    int *ret;
    unsigned long long *numbers;
    char *finalList = (char *) malloc(pitchSize * sizeof(char));
    char *final_words;

    hipMalloc((void **) &dev_words, pitchSize * sizeof(char));
    hipMemcpy(dev_words, list, pitchSize * sizeof(char), hipMemcpyHostToDevice);
    hipMalloc((void **) &ret, (*words) * sizeof(int));
    hipMemset(ret, 0, *words * sizeof(int));
    hipMalloc((void **) &numbers, (*words) * sizeof(unsigned long long int));
    hipMemset(numbers, 0, *words * sizeof(unsigned long long int));
    hipMalloc((void **) &final_words, pitchSize * sizeof(char));
    hipMemset(final_words, ' ', pitchSize * sizeof(char));

    free(list);

    int dimx;
    int dimy;
    if (pitchSize < 65535) {
        dimx = 1;
        dimy = *words; //wordsize
    }
    else {
        dimx = ceil(*words /(float)65535);
        dimy = 65535;
    }

    dim3 grid;
    grid.x = dimy;
    grid.y = dimx;

    kernel <<< grid, *wordLength, (*wordLength) * sizeof(char) >>>(dev_words, final_words, numbers, ret, *words, *wordLength, i);


    int * newArray = (int *)malloc(*words * sizeof(int));
    //int newArray[*words];
    hipFree(dev_words);
    hipMemcpy(newArray, ret, (*words) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(ret);
    hipFree(numbers);
    hipMemcpy(finalList, final_words, pitchSize * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(final_words);




    int k = 0;
    for(i = 0; i < *words; i++) {
        if(newArray[i] != 0) {
            k++;
        }
    }

    //build arrays
    int *finalOccur = (int *) malloc(k * sizeof(int));

    char **finalWords;
    finalWords = (char **) malloc(k * sizeof(char *));

    k = 0;
    for(i = 0; i < *words; i++) {
        if(newArray[i] != 0) {
            finalOccur[k] = newArray[i];

            memset(temp, 0, sizeof(temp));
            for(int j = 0; j < *wordLength; j++) {
                if(finalList[i*(*wordLength) + j] == ' ')
                    break;
                temp[j] = finalList[i*(*wordLength) + j];
            }

            finalWords[k] = (char *) malloc((strlen(temp) + 1) * sizeof(char));
            strcpy(finalWords[k], temp);

            k++;
        }
    }
    free(finalList);



    quick_sort(finalWords, finalOccur, 0, k - 1); //works!

    FILE *fout4;
    fout4 = fopen("cudaAlphaOutput.txt", "w");
    cudaSaveList(finalWords, finalOccur, k, fout4);
    fclose(fout4);

    quickSortOccur(finalOccur, finalWords, 0, k - 1); //works!

    FILE *fout3;
    fout3 = fopen("cudaOccurOutput.txt", "w");
    cudaSaveList(finalWords, finalOccur, k, fout3);
    fclose(fout3);



    //free last arrays
    free(finalOccur);
    for (int j = 0; j < k; j++) {
        free(finalWords[j]);
    }
    free(finalWords);

    timeb = currentTime();
    printf("\n\nParallel time spent: %lf seconds\n", timeb - timea);


    return 0;


}